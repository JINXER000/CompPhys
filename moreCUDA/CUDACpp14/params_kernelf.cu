/**
 * @file   : params_kernelf.cu
 * @brief  : Modified implementation of njuffa's; 
 * 				CUDA kernel functions as parameters with CUDA C++14, CUDA Unified Memory Management
 * @details : Modified implementation of njuffa's,  
 * 				std::function vs. function pointer in C++11, C++14, and now in CUDA
 * 				std::function vs. function pointer for CUDA kernel functions (i.e. __global__ )
 * 
 * @author : Ernest Yeung <ernestyalumni@gmail.com>
 * @date   : 20171020  
 * @ref    : https://devtalk.nvidia.com/default/topic/487190/kernel-functions-as-parameters-/
 * https://stackoverflow.com/questions/16498969/how-do-i-typedef-a-function-pointer-with-the-c11-using-syntax
 * 
 * https://www.paypal.com/cgi-bin/webscr?cmd=_donations&business=ernestsaveschristmas%2bpaypal%40gmail%2ecom&lc=US&item_name=ernestyalumni&currency_code=USD&bn=PP%2dDonationsBF%3abtn_donateCC_LG%2egif%3aNonHosted 
 * 
 * which won't go through a 3rd. party such as indiegogo, kickstarter, patreon.  
 * Otherwise, I receive emails and messages on how all my (free) material on 
 * physics, math, and engineering have helped students with their studies, 
 * and I know what it's like to not have money as a student, but love physics 
 * (or math, sciences, etc.), so I am committed to keeping all my material 
 * open-source and free, whether or not 
 * sufficiently crowdfunded, under the open-source MIT license: 
 * 	feel free to copy, edit, paste, make your own versions, share, use as you wish.  
 *  Just don't be an asshole and not give credit where credit is due.  
 * Peace out, never give up! -EY
 * 
 * */
/* 
 * COMPILATION TIP
 * nvcc -std=c++14 params_kernelf.cu -o params_kernelf.exe
 * 
 * */

#include <hip/hip_runtime.h>
#include <iostream>
#include <functional> 	// std::function
#include <type_traits> 	// std::add_pointer

#include <map>			// std::map


#include <stdio.h>

#include <vector> // std::vector

#include <memory> // std::unique_ptr

#define BLOCK_COUNT  240
#define THREAD_COUNT 128

/** @brief function pointer example, pf = "processing function"
 * @ref Scott Meyers, Effective Modern C++, pp. 63, Item 9
 * @details pointer to a function of 2 floats
 * */
// this WORKS
// using pf = float (*)(float, float);

using pf = std::add_pointer<float(float,float)>::type;

__device__ float minimum(float a, float b)
{
    return fminf(a,b);
};

__device__ float maximum(float a, float b)
{
    return fmaxf(a,b);
};

/**
 * @ref http://www.cplusplus.com/forum/beginner/4844/
 * 
 * @details From jsmith 
 * The ! operator is the logical-NOT operator.  
 * In C/C++, integers can be implicitly cast to boolean values according to the rule that 0 is false and anything else is true. The boolean false has "value" 0 and true has "value" 1.
 * Programmers sometimes take advantage of the implicit boolean conversion to perform the conversion:
 * f(x) = { 0, iff x == 0, and 1 iff x != 0 }
 * So given an integer x, !!x is 1 iff x is not zero and 0 otherwise.
 * 
 * */

/**
 * @brief 
 * @ref Scott Meyers, Effective Modern C++, pp. 63, Item 9
*/
__device__ pf func_d[2] = { maximum, minimum };


/* this will be inputted in with the following parameters  
 * x = x_d \in \mathbb{R}^n
 * res = res_d \in \mathbb{R}^{N_x}  
 * n = n = total length or size of the problem array  
 * pf func = func_d[findmin]
 */
__device__ void minmax(float *x, float *res, int n, pf func)
{
	int Mx = blockDim.x; // "THREAD_COUNT" or number of threads in a (single) block, in x-direction

//	__shared__ float partExtr[Mx];  // error: expression must have a constant value
 	__shared__ float partExtr[THREAD_COUNT];  // error: expression must have a constant value
  
    int i;	
    int tid = threadIdx.x; // 0,1,... Mx-1

	int kx = threadIdx.x + blockDim.x*blockIdx.x; // kx=0,1,.. Mx*Nx-1

	int Nx = gridDim.x ; 


    float extr = x[0];

	// i=kx, kx + Nx*Mx, kx + 2*Nx*Mx, ... < n 
	// the big idea is to make sure all values of x=x[i], i=0,1,...n-1, gets calculated
	// this is because we worry about the case when n > Nx*Mx (or perhaps n >> Nx*Mx !!!)
    for (i = kx; i < n; i += Nx*Mx) {
        extr = func(extr, x[i]);
    }
	// tid \in 0,1,...Mx-1; kx = tid + blockDim.x*blockIdx.x
    partExtr[tid] = extr;

	// Mx >> 1 = Mx/2; i= Mx/2, Mx/4, ... 2,1
    for (i = Mx >> 1; i > 0; i >>= 1) {
        __syncthreads(); 
		
		// tid \in 0,1,...Mx-1
		// assuming Mx % 2 = 0, 
		// i= Mx/2, Mx/4 ... 2,1
        if (tid < i) {
			// tid \in 0,1,...Mx-1
			// tid+i, so i < tid+i and Mx/2 < (tid + Mx/2), Mx/4 < (tid+ Mx/4) ... 1 < (tid +1)
			// i.e. we're grabbing the "other half", computing against the "other half" in a 
			// thread block, each time we're accessing tid+i
            partExtr[tid] = func(partExtr[tid], partExtr[tid+i]);
        }
    }

    if (tid == 0) {
        res[blockIdx.x] = partExtr[tid];
    }
}

__global__ void minmax_kernel(float *x, float *res, int n, int findmin)
{
    minmax(x, res, n, func_d[findmin]);
}

/**
 *	@fn findExtremum
 * 	@param int findmin - 0 for fmaxf, 1 for fminf
 * 
 * */
float findExtremum(float *x, int n, int findmin)
{
    pf func_h[2] = { fmaxf, fminf };

    float *res_d;	// result of minmax on device GPU; res_d \in \mathbb{R}^{ N_x }
    //float *res_h;	// result of minmax on host CPU
	std::vector<float> h_resultvec( BLOCK_COUNT, 0.f);  // h_resultvec \in \mathbb{R}^{ N_x}
    float *x_d;		// x array on device GPU, of size n, x \in \mathbb{R}^n
    float r;

    if (n < 1) return sqrtf(-1.0f); // NaN

    hipMalloc((void**)&res_d, BLOCK_COUNT*sizeof(res_d[0]));
    hipMalloc((void**)&x_d, n * sizeof(x_d[0]));

    hipMemcpy(x_d, x, n * sizeof(x_d[0]), hipMemcpyHostToDevice);

	// ! is logical NOT, and so !! makes a boolean, which is just 0 or 1
    minmax_kernel<<<BLOCK_COUNT,THREAD_COUNT>>>(x_d, res_d, n, !!findmin);
/*
    if (!res_h) {
		fprintf(stderr, "res_h allocation failed\n");
		exit(EXIT_FAILURE);
    }
*/
    hipMemcpy( h_resultvec.data(), res_d, BLOCK_COUNT * sizeof(res_d[0]), hipMemcpyDeviceToHost);

    hipFree(res_d);
    hipFree(x_d);

    r = h_resultvec[0];

	// i=1,2,...N_x-1; and h_resultvec \in \mathbb{R}^{N_x}
    for (int i = 1; i < BLOCK_COUNT; i++) {
		// int findmin =0,1; func_h[0]=fmaxf, func_h[1]=fminf
		r = func_h[findmin](r, h_resultvec[i]);
	}


    return r;
}

/* =============== sunburntfish's increment, decrement test function =============== */
// @ref https://stackoverflow.com/questions/25848690/should-i-use-stdfunction-or-a-function-pointer-in-c
// @ref pp. 39, Item 5 of Scott Meyers, Effective Modern C++  

// doesn't work, error: dynamic initialization not supported for __device__  
//using funcpf = std::function<float(float,float)>;

//  error: dynamic initialization is not supported for __device__, __constant__ and __shared__ variables.
//__device__ funcpf funcfunc_d[2] = { maximum, minimum };

/**
 * @ref https://stackoverflow.com/questions/16498969/how-do-i-typedef-a-function-pointer-with-the-c11-using-syntax
 * How do I typedef a function pointer with the C++11 using syntax?  answer from 0x499602D2
 * */
using pf1 = std::add_pointer<int(int)>::type;  

__device__ int increment_k(int t) {
	return t+ 1;
}

__device__ int decrement_k(int t) {
	return t-1;
}

/**
 * @fn incdecops
 * @ref Stanley B. Lippman, Josee Lajoie, Barbara E. Moo. C++ Primer 
 * */
//  error: dynamic initialization is not supported for __device__, __constant__ and __shared__ variables.
/* __device__ std::map<int, pf1> incdecops = {
	{0, increment_k}, 
	{1, decrement_k} };
*/

__device__ pf1 d_func[2] = { increment_k, decrement_k };

__global__ void test_k(int *testArray, int funcId, uint numPoints ) {
	unsigned int index = threadIdx.x + __umul24(blockIdx.x,blockDim.x); 
	
	if (index >= numPoints) { return ; }
	
	pf1 func = d_func[funcId];

	// i=kx, kx + Nx*Mx, kx + 2*Nx*Mx, ... < n 
	// the big idea is to make sure all values of x=x[i], i=0,1,...n-1, gets calculated
	// this is because we worry about the case when n > Nx*Mx (or perhaps n >> Nx*Mx !!!)
	int Mx = blockDim.x; 
	int Nx = gridDim.x; 
	for (int idx = index; idx < numPoints; idx += Nx*Mx) {
		int temp = func(testArray[idx]);
		testArray[idx] = temp;
	}
		
}

void test(int *testArray, int funcId, unsigned int numPoints, int numThreadsPerBlock) {
	// define grid and block size
	unsigned int numBlocks = (numPoints + numThreadsPerBlock -1)/numThreadsPerBlock;
//	computeGridSize(numPoints, 256, numBlocks, numThreadsPerBlock);

	test_k<<<numBlocks, numThreadsPerBlock>>>( testArray, funcId, numPoints); 
}

int main (void)
{
	// sanity check
	constexpr const int Mx = THREAD_COUNT;
	std::cout << " Mx >> 1 : " << ( Mx >> 1) << std::endl;
	for (int test_i = Mx >> 1; test_i >0; test_i >>= 1) { std::cout << test_i << " " ; } std::cout << std::endl;

	constexpr const int ELEM_COUNT = 8 ;
    float x[ELEM_COUNT] = {-1.3f, 2.4f, 3.5f, -2.3f, 4.5f, 0.4f, -5.3f, -1.6f};

    float minimum = findExtremum(x, ELEM_COUNT, 1);
    float maximum = findExtremum(x, ELEM_COUNT, 0);

    printf("min=% 13.6e  max=% 13.6e\n", minimum, maximum);

	std::cout << std::endl << "increment, decrement " << std::endl ;

	int numPoints = 10;
	
	// on host
	std::vector<int> testArray(numPoints,0);
	for (uint i=0; i<numPoints;i++) { testArray[i] = i; }
	
	auto deleterZZ_lambda=[&](int* ptr){ hipFree(ptr); };
	std::unique_ptr<int[], decltype(deleterZZ_lambda)> d_test(new int[numPoints], deleterZZ_lambda); // u for unique and unified
	hipMallocManaged((void **) &d_test, numPoints*sizeof(float));

	std::cout << " Before : " << std::endl;
	for (auto ele : testArray) { std::cout << ele << " " ; } std::cout << std::endl;

	hipMemcpy(d_test.get(),testArray.data(),numPoints*sizeof(int),hipMemcpyHostToDevice); 
	
	test(d_test.get(), 0, numPoints, 16);
	
	hipMemcpy(testArray.data(),d_test.get(), numPoints*sizeof(int),hipMemcpyDeviceToHost); 
	 
	std::cout << " After : " << std::endl; 
	for (auto ele : testArray) { std::cout << ele << " " ; } std::cout << std::endl;


    return EXIT_SUCCESS;
}

